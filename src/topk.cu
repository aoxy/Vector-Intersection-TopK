#include "hip/hip_runtime.h"

#include "topk.h"
#include <thread>
#include <cassert>

typedef uint4 group_t; // uint32_t

void __global__ docQueryScoringCoalescedMemoryAccessSampleKernel(
        const __restrict__ uint16_t *docs, 
        const int *doc_lens, const size_t n_docs, 
        uint16_t *query, const int query_len, float *scores) {
    // each thread process one doc-query pair scoring task
    register auto tid = blockIdx.x * blockDim.x + threadIdx.x, tnum = gridDim.x * blockDim.x;

    if (tid >= n_docs) {
        return;
    }

    __shared__ uint16_t query_on_shm[MAX_QUERY_SIZE];
#pragma unroll
    for (auto i = threadIdx.x; i < query_len; i += blockDim.x) {
        query_on_shm[i] = query[i]; // not very efficient query loading temporally, as assuming its not hotspot
    }

    __syncthreads();

    for (auto doc_id = tid; doc_id < n_docs; doc_id += tnum) {
        register int query_idx = 0;

        register float tmp_score = 0.;

        register bool no_more_load = false;

        for (auto i = 0; i < MAX_DOC_SIZE / (sizeof(group_t) / sizeof(uint16_t)); i++) {
            if (no_more_load) {
                break;
            }
            register group_t loaded = ((group_t *)docs)[i * n_docs + doc_id]; // tid
            register uint16_t *doc_segment = (uint16_t*)(&loaded);
            for (auto j = 0; j < sizeof(group_t) / sizeof(uint16_t); j++) {
                if (doc_segment[j] == 0) {
                    no_more_load = true;
                    break;
                    // return;
                }
                while (query_idx < query_len && query_on_shm[query_idx] < doc_segment[j]) {
                    ++query_idx;
                }
                if (query_idx < query_len) {
                    tmp_score += (query_on_shm[query_idx] == doc_segment[j]);
                }
            }
            __syncwarp();
        }
        scores[doc_id] = tmp_score / max(query_len, doc_lens[doc_id]); // tid
    }
}

void doc_query_scoring_gpu_function(std::vector<std::vector<uint16_t>> &querys,
    std::vector<std::vector<uint16_t>> &docs,
    std::vector<uint16_t> &lens,
    std::vector<std::vector<int>> &indices //shape [querys.size(), TOPK]
    ) {

    auto n_docs = docs.size();
    std::vector<float> scores(n_docs);
    std::vector<int> s_indices(n_docs);

    float *d_scores = nullptr;
    uint16_t *d_docs = nullptr, *d_query = nullptr;
    int *d_doc_lens = nullptr;

    // copy to device
    hipMalloc(&d_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    hipMalloc(&d_scores, sizeof(float) * n_docs);
    hipMalloc(&d_doc_lens, sizeof(int) * n_docs);

    uint16_t *h_docs = new uint16_t[MAX_DOC_SIZE * n_docs];
    memset(h_docs, 0, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs);
    std::vector<int> h_doc_lens_vec(n_docs);
    for (int i = 0; i < docs.size(); i++) {
        for (int j = 0; j < docs[i].size(); j++) {
            auto group_sz = sizeof(group_t) / sizeof(uint16_t);
            auto layer_0_offset = j / group_sz;
            auto layer_0_stride = n_docs * group_sz;
            auto layer_1_offset = i;
            auto layer_1_stride = group_sz;
            auto layer_2_offset = j % group_sz;
            auto final_offset = layer_0_offset * layer_0_stride + layer_1_offset * layer_1_stride + layer_2_offset;
            h_docs[final_offset] = docs[i][j];
        }
        h_doc_lens_vec[i] = docs[i].size();
    }

    hipMemcpy(d_docs, h_docs, sizeof(uint16_t) * MAX_DOC_SIZE * n_docs, hipMemcpyHostToDevice);
    hipMemcpy(d_doc_lens, h_doc_lens_vec.data(), sizeof(int) * n_docs, hipMemcpyHostToDevice);

    hipDeviceProp_t device_props;
    hipGetDeviceProperties(&device_props, 0);

    hipSetDevice(0);

    for(auto& query : querys) {
        //init indices
        for (int i = 0; i < n_docs; ++i) {
            s_indices[i] = i;
        }

        const size_t query_len = query.size();
        hipMalloc(&d_query, sizeof(uint16_t) * query_len);
        hipMemcpy(d_query, query.data(), sizeof(uint16_t) * query_len, hipMemcpyHostToDevice);

        // launch kernel
        int block = N_THREADS_IN_ONE_BLOCK;
        int grid = (n_docs + block - 1) / block;
        docQueryScoringCoalescedMemoryAccessSampleKernel<<<grid, block>>>(d_docs,
            d_doc_lens, n_docs, d_query, query_len, d_scores);
        hipDeviceSynchronize();

        hipMemcpy(scores.data(), d_scores, sizeof(float) * n_docs, hipMemcpyDeviceToHost);

        // sort scores
        std::partial_sort(s_indices.begin(), s_indices.begin() + TOPK, s_indices.end(),
                        [&scores](const int& a, const int& b) {
                            if (scores[a] != scores[b]) {
                                return scores[a] > scores[b];  // 按照分数降序排序
                            }
                            return a < b;  // 如果分数相同，按索引从小到大排序
                    });
        std::vector<int> s_ans(s_indices.begin(), s_indices.begin() + TOPK);
        indices.push_back(s_ans);

        hipFree(d_query);
    }

    // deallocation
    hipFree(d_docs);
    //hipFree(d_query);
    hipFree(d_scores);
    hipFree(d_doc_lens);
    free(h_docs);

}

void do_doc_query_scoring(const std::vector<std::vector<uint16_t>> &docs,
                          const std::vector<std::vector<unsigned char>> &querys_map,
                          const std::vector<size_t> &querys_len,
                          std::vector<std::vector<float>> &scores,
                          const size_t from, const size_t to)
{

    for (size_t q = 0; q < querys_map.size(); q++)
    {
        for (size_t d = from; d < to; d++)
        {
            uint16_t inter = 0;
            for (const uint16_t id : docs[d])
            {
                inter += querys_map[q][id];
            }
            scores[q][d] = inter * 1.0 / std::max(querys_len[q], docs[d].size());
        }
    }
}

void do_scoring_topk(const std::vector<std::vector<float>> &scores, const std::vector<int> &s_indices, std::vector<std::vector<int>> &indices, const size_t from, const size_t to)
{
    for (size_t q = from; q < to; q++)
    {
        std::vector<int> new_indices = s_indices;
        const std::vector<float> &query_scores = scores[q];
        std::partial_sort(new_indices.begin(), new_indices.begin() + TOPK, new_indices.end(),
                          [&query_scores](const int &a, const int &b)
                          {
                              if (query_scores[a] != query_scores[b])
                              {
                                  return query_scores[a] > query_scores[b];
                              }
                              return a < b;
                          });
        std::vector<int> s_ans(new_indices.begin(), new_indices.begin() + TOPK);
        indices[q].swap(s_ans);
    }
}

void doc_query_scoring_cpu_function(std::vector<std::vector<uint16_t>> &querys,
                                    std::vector<std::vector<uint16_t>> &docs,
                                    std::vector<std::vector<int>> &indices // shape [querys.size(), TOPK]
)
{

    size_t n_docs = docs.size();
    size_t n_querys = querys.size();
    size_t n_threads = N_THREADS_CPU;
    if (n_threads > n_docs)
    {
        n_threads = n_docs;
    }
    size_t n_docs_per_thread = n_docs / n_threads;
    size_t n_onemore_doc_thread = n_docs - n_docs_per_thread * n_threads;
    std::vector<size_t> docs_from(n_threads);
    std::vector<size_t> docs_to(n_threads);
    for (size_t i = 0; i < n_threads; i++)
    {
        if (i < n_onemore_doc_thread)
        {
            docs_from[i] = i * (n_docs_per_thread + 1);
            docs_to[i] = (i + 1) * (n_docs_per_thread + 1);
        }
        else
        {
            docs_from[i] = i * (n_docs_per_thread) + n_onemore_doc_thread;
            docs_to[i] = (i + 1) * (n_docs_per_thread) + n_onemore_doc_thread;
        }
    }

    std::vector<std::vector<float>> scores(n_querys, std::vector<float>(n_docs, 0.0));
    std::vector<std::vector<unsigned char>> querys_map(n_querys, std::vector<unsigned char>(MAX_ID, 0));
    std::vector<size_t> querys_len(n_querys);
    for (size_t q = 0; q < querys.size(); q++)
    {
        for (const uint16_t &id : querys[q])
        {
            querys_map[q][id] = 1;
        }
        querys_len[q] = querys[q].size();
    }

    std::vector<std::thread> scoring_threads(n_threads - 1);
    for (size_t i = 0; i < n_threads - 1; i++)
    {
        scoring_threads[i] = std::thread([&, i]()
                                         { do_doc_query_scoring(docs, querys_map, querys_len, scores, docs_from[i], docs_to[i]); });
    }
    do_doc_query_scoring(docs, querys_map, querys_len, scores, docs_from[n_threads - 1], docs_to[n_threads - 1]);
    for (auto &t : scoring_threads)
    {
        t.join();
    }

    std::cout << "scoring ok!" << std::endl;

    // Top K
    indices.resize(n_querys);
    n_threads = N_THREADS_CPU;
    if (n_threads > n_querys)
    {
        n_threads = n_querys;
    }
    size_t n_querys_per_thread = n_querys / n_threads;
    size_t n_onemore_query_thread = n_querys - n_querys_per_thread * n_threads;
    std::vector<size_t> querys_from(n_threads);
    std::vector<size_t> querys_to(n_threads);
    for (size_t i = 0; i < n_threads; i++)
    {
        if (i < n_onemore_query_thread)
        {
            querys_from[i] = i * (n_querys_per_thread + 1);
            querys_to[i] = (i + 1) * (n_querys_per_thread + 1);
        }
        else
        {
            querys_from[i] = i * (n_querys_per_thread) + n_onemore_query_thread;
            querys_to[i] = (i + 1) * (n_querys_per_thread) + n_onemore_query_thread;
        }
    }

    std::vector<int> s_indices(n_docs);
    for (int i = 0; i < n_docs; ++i)
    {
        s_indices[i] = i;
    }
    std::vector<std::thread> sorting_threads(n_threads - 1);
    for (size_t i = 0; i < n_threads - 1; i++)
    {
        sorting_threads[i] = std::thread([&, i]()
                                         { do_scoring_topk(scores, s_indices, indices, querys_from[i], querys_to[i]); });
    }
    do_scoring_topk(scores, s_indices, indices, querys_from[n_threads - 1], querys_to[n_threads - 1]);
    for (auto &t : sorting_threads)
    {
        t.join();
    }
}

int compare(std::vector<std::vector<int>> &indices_1, std::vector<std::vector<int>> &indices_2) {
    assert(indices_1.size() == indices_2.size());
    for (int i = 0; i < indices_1.size(); i++) {
        assert(indices_1[i].size() == indices_2[i].size());
        for (int j = 0; j < indices_1[i].size(); j++) {
            if (indices_1[i][j] != indices_2[i][j]) {
                printf("r=%d, c=%d, indices(%d) != indices_baseline(%d)\n", i, j, indices_1[i][j], indices_2[i][j]);
                return 0;
            }
        }
    }
    printf("compare done!\n");

    return 0;
}